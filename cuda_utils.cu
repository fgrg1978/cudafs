#include <stdio.h>
#include <string.h>
#include <errno.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "config.h"

extern "C" {
#include "cuda_utils.h"
}


extern "C" int gpu_write(struct metadata *file, struct list *flist)
{
	hipError_t err;

	if (file[flist->inode].cuda_state & LOCK_FILE)
		return 0;

	if (file[flist->inode].cuda_data) 
		hipFree(file[flist->inode].cuda_data);

	err = hipMalloc(&file[flist->inode].cuda_data, file[flist->inode].size-1);
	if(err != hipSuccess) {
		return 0;
	} else {
		file[flist->inode].cuda_state |= LOCK_FILE;

		err = hipMemcpy(file[flist->inode].cuda_data,
				file[flist->inode].data, 
				file[flist->inode].size,
				hipMemcpyHostToDevice);
		if(err != hipSuccess) {
			file[flist->inode].cuda_state = ~LOCK_FILE;
			return 0;
		} else {
			file[flist->inode].cuda_state = 0;
			file [flist->inode].cuda_state |= IN_CUDA_MEM;
			free(file [flist->inode].data);
		}
	}
	return 1;
}


extern "C" int gpu_read(struct metadata *file, struct list *flist, int free)
{
	hipError_t err;

	if (file[flist->inode].cuda_state & LOCK_FILE || !(file[flist->inode].cuda_state & IN_CUDA_MEM) || file[flist->inode].data) 
		return 0;

	if (file[flist->inode].cuda_state & IN_CUDA_MEM &&
			(!file[flist->inode].data)) 
	{
		file[flist->inode].data = (char *) malloc( file[flist->inode].size);

		if (!file[flist->inode].data) return -ENOMEM;

		err = hipMemcpy(file[flist->inode].data,
				file[flist->inode].cuda_data,
				file[flist->inode].size-1,
				hipMemcpyDeviceToHost);

		if(err != hipSuccess)
			return -ENOMEM;
		
		file [flist->inode].cuda_state = 0;
		
		if (free) 
			hipFree(file[flist->inode].cuda_data);
		else
			file [flist->inode].cuda_state |= IN_CUDA_MEM;
	}
	return 1;
}

extern "C" void gpu_free(struct metadata *file, struct list *flist)
{

	if (file[flist->inode].cuda_state & LOCK_FILE) return;

	if (file[flist->inode].cuda_data) hipFree(file[flist->inode].cuda_data);
}
